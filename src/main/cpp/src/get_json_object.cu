#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "get_json_object.hpp"

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/offsets_iterator_factory.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/json/json.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

namespace spark_rapids_jni {

namespace detail {

/**
 * @brief Result of calling a parse function.
 *
 * The primary use of this is to distinguish between "success" and
 * "success but no data" return cases.  For example, if you are reading the
 * values of an array you might call a parse function in a while loop. You
 * would want to continue doing this until you either encounter an error
 * (parse_result::ERROR) or you get nothing back (parse_result::EMPTY)
 */
enum class parse_result { ERROR, SUCCESS };

CUDF_HOST_DEVICE inline bool path_is_empty(size_t path_size) { return path_size == 0; }

CUDF_HOST_DEVICE inline bool path_match_element(path_instruction const* path_ptr,
                                                size_t path_size,
                                                path_instruction_type path_type0)
{
  if (path_size < 1) { return false; }
  return path_ptr[0].type == path_type0;
}

CUDF_HOST_DEVICE inline bool path_match_elements(path_instruction const* path_ptr,
                                                 size_t path_size,
                                                 path_instruction_type path_type0,
                                                 path_instruction_type path_type1)
{
  if (path_size < 2) { return false; }
  return path_ptr[0].type == path_type0 && path_ptr[1].type == path_type1;
}

CUDF_HOST_DEVICE inline bool path_match_elements(path_instruction const* path_ptr,
                                                 size_t path_size,
                                                 path_instruction_type path_type0,
                                                 path_instruction_type path_type1,
                                                 path_instruction_type path_type2,
                                                 path_instruction_type path_type3)
{
  if (path_size < 4) { return false; }
  return path_ptr[0].type == path_type0 && path_ptr[1].type == path_type1 &&
         path_ptr[2].type == path_type2 && path_ptr[3].type == path_type3;
}

CUDF_HOST_DEVICE inline thrust::tuple<bool, int> path_match_subscript_index(
  path_instruction const* path_ptr, size_t path_size)
{
  auto match = path_match_elements(
    path_ptr, path_size, path_instruction_type::subscript, path_instruction_type::index);
  if (match) {
    return thrust::make_tuple(true, path_ptr[1].index);
  } else {
    return thrust::make_tuple(false, 0);
  }
}

CUDF_HOST_DEVICE inline thrust::tuple<bool, cudf::string_view> path_match_named(
  path_instruction const* path_ptr, size_t path_size)
{
  auto match = path_match_element(path_ptr, path_size, path_instruction_type::named);
  if (match) {
    return thrust::make_tuple(true, path_ptr[0].name);
  } else {
    return thrust::make_tuple(false, cudf::string_view());
  }
}

CUDF_HOST_DEVICE inline thrust::tuple<bool, int> path_match_subscript_index_subscript_wildcard(
  path_instruction const* path_ptr, size_t path_size)
{
  auto match = path_match_elements(path_ptr,
                                   path_size,
                                   path_instruction_type::subscript,
                                   path_instruction_type::index,
                                   path_instruction_type::subscript,
                                   path_instruction_type::wildcard);
  if (match) {
    return thrust::make_tuple(true, path_ptr[1].index);
  } else {
    return thrust::make_tuple(false, 0);
  }
}

template <int max_json_nesting_depth>
CUDF_HOST_DEVICE bool evaluate_path(json_parser<max_json_nesting_depth>& p,
                                    json_generator<max_json_nesting_depth>& g,
                                    bool g_contains_outer_array_pairs,
                                    write_style style,
                                    path_instruction const* path_ptr,
                                    int path_size)
{
  auto token = p.get_current_token();

  // case (VALUE_STRING, Nil) if style == RawStyle
  if (json_token::VALUE_STRING == token && path_is_empty(path_size) &&
      style == write_style::raw_style) {
    // there is no array wildcard or slice parent, emit this string without
    // quotes write current string in parser to generator
    g.write_raw(p);
    return true;
  }
  // case (START_ARRAY, Nil) if style == FlattenStyle
  else if (json_token::START_ARRAY == token && path_is_empty(path_size) &&
           style == write_style::flatten_style) {
    // flatten this array into the parent
    bool dirty = false;
    while (json_token::END_ARRAY != p.next_token()) {
      // JSON validation check
      if (json_token::ERROR == p.get_current_token()) { return false; }

      dirty |= evaluate_path(p, g, true, style, nullptr, 0);
    }
    return dirty;
  }
  // case (_, Nil)
  else if (path_is_empty(path_size)) {
    // general case: just copy the child tree verbatim
    return g.copy_current_structure(p);
  }
  // case (START_OBJECT, Key :: xs)
  else if (json_token::START_OBJECT == token &&
           path_match_element(path_ptr, path_size, path_instruction_type::key)) {
    bool dirty = false;
    while (json_token::END_OBJECT != p.next_token()) {
      // JSON validation check
      if (json_token::ERROR == p.get_current_token()) { return false; }

      if (dirty) {
        // once a match has been found we can skip other fields
        if (!p.try_skip_children()) {
          // JSON validation check
          return false;
        }
      } else {
        dirty = evaluate_path(p, g, true, style, path_ptr + 1, path_size - 1);
      }
    }
    return dirty;
  }
  // case (START_ARRAY, Subscript :: Wildcard :: Subscript :: Wildcard :: xs)
  else if (json_token::START_ARRAY == token &&
           path_match_elements(path_ptr,
                               path_size,
                               path_instruction_type::subscript,
                               path_instruction_type::wildcard,
                               path_instruction_type::subscript,
                               path_instruction_type::wildcard)) {
    // special handling for the non-structure preserving double wildcard
    // behavior in Hive
    bool dirty = false;
    g.write_start_array();
    while (p.next_token() != json_token::END_ARRAY) {
      // JSON validation check
      if (json_token::ERROR == p.get_current_token()) { return false; }

      dirty |= evaluate_path(p, g, true, write_style::flatten_style, path_ptr + 4, path_size - 4);
    }
    g.write_end_array();
    return dirty;
  }
  // case (START_ARRAY, Subscript :: Wildcard :: xs) if style != QuotedStyle
  else if (json_token::START_ARRAY == token &&
           path_match_elements(path_ptr,
                               path_size,
                               path_instruction_type::subscript,
                               path_instruction_type::wildcard) &&
           style != write_style::quoted_style) {
    // retain Flatten, otherwise use Quoted... cannot use Raw within an array
    write_style next_style;
    switch (style) {
      case write_style::raw_style: next_style = write_style::quoted_style; break;
      case write_style::flatten_style: next_style = write_style::flatten_style;
    }

    // temporarily buffer child matches, the emitted json will need to be
    // modified slightly if there is only a single element written

    int dirty    = 0;
    auto child_g = g.new_child_generator();

    // child generator write a fake start array
    child_g.write_start_array_fake();

    while (p.next_token() != json_token::END_ARRAY) {
      // JSON validation check
      if (json_token::ERROR == p.get_current_token()) { return false; }

      // track the number of array elements and only emit an outer array if
      // we've written more than one element, this matches Hive's behavior
      dirty += (evaluate_path(p, child_g, false, next_style, path_ptr + 2, path_size - 2) ? 1 : 0);
    }

    // child generator write a fake end array
    child_g.write_end_array_fake();

    char* child_g_start = child_g.get_output_start_position();
    size_t child_g_len  = child_g.get_output_len() - 2;  // exclude [ ]

    if (dirty > 1) {
      // add outer array tokens
      g.write_child_raw_value(child_g_start, child_g_len, true);
    } else if (dirty == 1) {
      // remove outer array tokens
      g.write_child_raw_value(child_g_start, child_g_len, false);
    }  // else do not write anything

    return dirty > 0;
  }
  // case (START_ARRAY, Subscript :: Wildcard :: xs)
  else if (json_token::START_ARRAY == token &&
           path_match_elements(path_ptr,
                               path_size,
                               path_instruction_type::subscript,
                               path_instruction_type::wildcard)) {
    bool dirty = false;
    g.write_start_array();
    while (p.next_token() != json_token::END_ARRAY) {
      // JSON validation check
      if (json_token::ERROR == p.get_current_token()) { return false; }

      // wildcards can have multiple matches, continually update the dirty count
      dirty |= evaluate_path(p, g, true, write_style::quoted_style, path_ptr + 2, path_size - 2);
    }
    g.write_end_array();

    return dirty;
  }
  // case (START_ARRAY, Subscript :: Index(idx) :: (xs@Subscript :: Wildcard ::
  // _))
  else if (json_token::START_ARRAY == token &&
           thrust::get<0>(path_match_subscript_index_subscript_wildcard(path_ptr, path_size))) {
    int idx = thrust::get<1>(path_match_subscript_index_subscript_wildcard(path_ptr, path_size));
    p.next_token();
    // JSON validation check
    if (json_token::ERROR == p.get_current_token()) { return false; }

    int i = idx;
    while (i >= 0) {
      if (p.get_current_token() == json_token::END_ARRAY) {
        // terminate, nothing has been written
        return false;
      }
      if (0 == i) {
        bool dirty =
          evaluate_path(p, g, true, write_style::quoted_style, path_ptr + 2, path_size - 2);
        while (p.next_token() != json_token::END_ARRAY) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return false; }

          // advance the token stream to the end of the array
          if (!p.try_skip_children()) { return false; }
        }
        return dirty;
      } else {
        // i > 0
        if (!p.try_skip_children()) { return false; }

        p.next_token();
        // JSON validation check
        if (json_token::ERROR == p.get_current_token()) { return false; }
      }
      --i;
    }
    // path parser guarantees idx >= 0
    // will never reach to here
    return false;
  }
  // case (START_ARRAY, Subscript :: Index(idx) :: xs)
  else if (json_token::START_ARRAY == token &&
           thrust::get<0>(path_match_subscript_index(path_ptr, path_size))) {
    int idx = thrust::get<1>(path_match_subscript_index(path_ptr, path_size));
    p.next_token();
    // JSON validation check
    if (json_token::ERROR == p.get_current_token()) { return false; }

    int i = idx;
    while (i >= 0) {
      if (p.get_current_token() == json_token::END_ARRAY) {
        // terminate, nothing has been written
        return false;
      }
      if (0 == i) {
        bool dirty = evaluate_path(p, g, true, style, path_ptr + 2, path_size - 2);
        while (p.next_token() != json_token::END_ARRAY) {
          // JSON validation check
          if (json_token::ERROR == p.get_current_token()) { return false; }

          // advance the token stream to the end of the array
          if (!p.try_skip_children()) { return false; }
        }
        return dirty;
      } else {
        // i > 0
        if (!p.try_skip_children()) { return false; }

        p.next_token();
        // JSON validation check
        if (json_token::ERROR == p.get_current_token()) { return false; }
      }
      --i;
    }
    // path parser guarantees idx >= 0
    // will never reach to here
    return false;
  }
  // case (FIELD_NAME, Named(name) :: xs) if p.getCurrentName == name
  else if (json_token::FIELD_NAME == token &&
           thrust::get<0>(path_match_named(path_ptr, path_size)) &&
           p.match_current_field_name(thrust::get<1>(path_match_named(path_ptr, path_size)))) {
    if (p.next_token() != json_token::VALUE_NULL) {
      // JSON validation check
      if (json_token::ERROR == p.get_current_token()) { return false; }

      return evaluate_path(p, g, true, style, path_ptr + 1, path_size - 1);
    } else {
      return false;
    }
  }
  // case (FIELD_NAME, Wildcard :: xs)
  else if (json_token::FIELD_NAME == token &&
           path_match_element(path_ptr, path_size, path_instruction_type::wildcard)) {
    p.next_token();
    // JSON validation check
    if (json_token::ERROR == p.get_current_token()) { return false; }

    return evaluate_path(p, g, true, style, path_ptr + 1, path_size - 1);
    // case _ =>
  } else {
    if (!p.try_skip_children()) { return false; }
    return false;
  }
}

/**
 * @brief Parse a single json string using the provided command buffer
 *
 * @param j_parser The incoming json string and associated parser
 * @param path_ptr The command buffer to be applied to the string.
 * @param path_size Command buffer size
 * @param output Buffer used to store the results of the query
 * @returns A result code indicating success/fail/empty.
 */
template <int max_json_nesting_depth = curr_max_json_nesting_depth>
__device__ parse_result parse_json_path(json_parser<max_json_nesting_depth>& j_parser,
                                        path_instruction const* path_ptr,
                                        size_t path_size,
                                        json_generator<max_json_nesting_depth>& output)
{
  j_parser.next_token();
  // JSON validation check
  if (json_token::ERROR == j_parser.get_current_token()) { return parse_result::ERROR; }

  auto matched = evaluate_path(j_parser, output, true, write_style::raw_style, path_ptr, path_size);
  return matched ? parse_result::SUCCESS : parse_result::ERROR;
}

/**
 * @brief Parse a single json string using the provided command buffer
 *
 * This function exists primarily as a shim for debugging purposes.
 *
 * @param input The incoming json string
 * @param input_len Size of the incoming json string
 * @param commands The command buffer to be applied to the string. Always ends
 * with a path_operator_type::END
 * @param out_buf Buffer user to store the results of the query (nullptr in the
 * size computation step)
 * @param out_buf_size Size of the output buffer
 * @param options Options controlling behavior
 * @returns A pair containing the result code the output buffer.
 */
template <int max_json_nesting_depth = curr_max_json_nesting_depth>
__device__ thrust::pair<parse_result, json_generator<max_json_nesting_depth>>
get_json_object_single(
  char const* input,
  cudf::size_type input_len,
  path_instruction const* path_commands_ptr,
  int path_commands_size,
  char* out_buf,
  size_t out_buf_size,
  json_parser_options options)  // TODO make this a reference? use a global singleton options?
                                // reduce the copy contructor overhead
{
  json_parser j_parser(options, input, input_len);
  json_generator generator(out_buf, out_buf_size);
  auto const result = parse_json_path(j_parser, path_commands_ptr, path_commands_size, generator);
  return {result, generator};
}

/**
 * @brief Kernel for running the JSONPath query.
 *
 * This kernel operates in a 2-pass way.  On the first pass, it computes
 * output sizes.  On the second pass it fills in the provided output buffers
 * (chars and validity)
 *
 * @param col Device view of the incoming string
 * @param commands JSONPath command buffer
 * @param output_offsets Buffer used to store the string offsets for the results
 * of the query
 * @param out_buf Buffer used to store the results of the query
 * @param out_validity Output validity buffer
 * @param out_valid_count Output count of # of valid bits
 * @param options Options controlling behavior
 */
template <int block_size>
__launch_bounds__(block_size) CUDF_KERNEL
  void get_json_object_kernel(cudf::column_device_view col,
                              path_instruction const* path_commands_ptr,
                              int path_commands_size,
                              cudf::size_type* d_sizes,
                              cudf::detail::input_offsetalator output_offsets,
                              thrust::optional<char*> out_buf,
                              thrust::optional<cudf::bitmask_type*> out_validity,
                              thrust::optional<cudf::size_type*> out_valid_count,
                              json_parser_options options)
{
  auto tid          = cudf::detail::grid_1d::global_thread_id();
  auto const stride = cudf::thread_index_type{blockDim.x} * cudf::thread_index_type{gridDim.x};

  cudf::size_type warp_valid_count{0};

  auto active_threads = __ballot_sync(0xffff'ffffu, tid < col.size());
  while (tid < col.size()) {
    bool is_valid               = false;
    cudf::string_view const str = col.element<cudf::string_view>(tid);
    cudf::size_type output_size = 0;
    if (str.size_bytes() > 0) {
      char* dst = out_buf.has_value() ? out_buf.value() + output_offsets[tid] : nullptr;
      size_t const dst_size =
        out_buf.has_value() ? output_offsets[tid + 1] - output_offsets[tid] : 0;

      // process one single row
      auto [result, out] = get_json_object_single(str.data(),
                                                  str.size_bytes(),
                                                  path_commands_ptr,
                                                  path_commands_size,
                                                  dst,
                                                  dst_size,
                                                  options);
      output_size        = out.get_output_len();
      if (result == parse_result::SUCCESS) { is_valid = true; }
    }

    // filled in only during the precompute step. during the compute step, the
    // offsets are fed back in so we do -not- want to write them out
    if (!out_buf.has_value()) { d_sizes[tid] = output_size; }

    // validity filled in only during the output step
    if (out_validity.has_value()) {
      uint32_t mask = __ballot_sync(active_threads, is_valid);
      // 0th lane of the warp writes the validity
      if (!(tid % cudf::detail::warp_size)) {
        out_validity.value()[cudf::word_index(tid)] = mask;
        warp_valid_count += __popc(mask);
      }
    }

    tid += stride;
    active_threads = __ballot_sync(active_threads, tid < col.size());
  }

  // sum the valid counts across the whole block
  if (out_valid_count) {
    cudf::size_type block_valid_count =
      cudf::detail::single_lane_block_sum_reduce<block_size, 0>(warp_valid_count);
    if (threadIdx.x == 0) { atomicAdd(out_valid_count.value(), block_valid_count); }
  }
}

std::unique_ptr<cudf::column> get_json_object(cudf::strings_column_view const& col,
                                              cudf::string_scalar const& json_path,
                                              rmm::cuda_stream_view stream,
                                              rmm::mr::device_memory_resource* mr)
{
  if (col.is_empty()) return cudf::make_empty_column(cudf::type_id::STRING);

  // parse the json_path into a command buffer
  auto path_commands_optional = parse_path(json_path);

  // if the json path is empty, return a string column containing all nulls
  if (!path_commands_optional.has_value()) {
    return std::make_unique<cudf::column>(
      cudf::data_type{cudf::type_id::STRING},
      col.size(),
      // no data
      rmm::device_buffer{0, stream, mr},
      cudf::detail::create_null_mask(col.size(), cudf::mask_state::ALL_NULL, stream, mr),
      // null count
      col.size());
  }

  // compute output sizes
  auto sizes = rmm::device_uvector<cudf::size_type>(
    col.size(), stream, rmm::mr::get_current_device_resource());
  auto d_offsets = cudf::detail::offsetalator_factory::make_input_iterator(col.offsets());

  constexpr int block_size = 512;
  cudf::detail::grid_1d const grid{col.size(), block_size};
  auto cdv = cudf::column_device_view::create(col.parent(), stream);

  // create json parser options
  spark_rapids_jni::json_parser_options options;
  options.set_allow_single_quotes(true);
  options.set_allow_unescaped_control_chars(true);
  options.set_max_string_len(true);
  options.set_max_num_len(true);
  options.set_allow_tailing_sub_string(true);

  // preprocess sizes (returned in the offsets buffer)
  get_json_object_kernel<block_size>
    <<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
      *cdv,
      path_commands_optional.value().data(),
      path_commands_optional.value().size(),
      sizes.data(),
      d_offsets,
      thrust::nullopt,
      thrust::nullopt,
      thrust::nullopt,
      options);

  // convert sizes to offsets
  auto [offsets, output_size] =
    cudf::strings::detail::make_offsets_child_column(sizes.begin(), sizes.end(), stream, mr);
  d_offsets = cudf::detail::offsetalator_factory::make_input_iterator(offsets->view());

  // allocate output string column
  rmm::device_uvector<char> chars(output_size, stream, mr);

  // potential optimization : if we know that all outputs are valid, we could
  // skip creating the validity mask altogether
  rmm::device_buffer validity =
    cudf::detail::create_null_mask(col.size(), cudf::mask_state::UNINITIALIZED, stream, mr);

  // compute results
  rmm::device_scalar<cudf::size_type> d_valid_count{0, stream};

  get_json_object_kernel<block_size>
    <<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
      *cdv,
      path_commands_optional.value().data(),
      path_commands_optional.value().size(),
      sizes.data(),
      d_offsets,
      chars.data(),
      static_cast<cudf::bitmask_type*>(validity.data()),
      d_valid_count.data(),
      options);

  auto result = make_strings_column(col.size(),
                                    std::move(offsets),
                                    chars.release(),
                                    col.size() - d_valid_count.value(stream),
                                    std::move(validity));
  // unmatched array query may result in unsanitized '[' value in the result
  if (cudf::detail::has_nonempty_nulls(result->view(), stream)) {
    result = cudf::detail::purge_nonempty_nulls(result->view(), stream, mr);
  }
  return result;
}

}  // namespace detail

std::unique_ptr<cudf::column> get_json_object(cudf::strings_column_view const& col,
                                              cudf::string_scalar const& json_path,
                                              rmm::cuda_stream_view stream,
                                              rmm::mr::device_memory_resource* mr)
{
  // TODO: here do not know if json path is invalid, should handle it in Plugin
  return detail::get_json_object(col, json_path, stream, mr);
}

}  // namespace spark_rapids_jni
